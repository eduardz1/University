// -----------------------------------------------------------------------------
// * Name:       main_gpu.cxx
// * Purpose:    Driver for matrix multiplication on GPU
// * History:    Christophe Picard, Fall 2021
// -----------------------------------------------------------------------------

// includes, system
#include <cmath>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

// Parsing command line options using cxxopts 
// https://github.com/jarro2783/cxxopts.git
#include "args.hxx"

// Matrix manipulation function
#include "matrix_utils.h"

// Define different gemm kernel
#include <gemm_kernel.cuh>


#define REAL float
#define BLOCK_SIZE 32

///
/// Top level driver
///
int main(int argc, char **argv) {

  std::cout << "[Matrix Multiply Using CUDA] - Starting..." << std::endl;

  // Define parser 
  args::ArgumentParser parser("gemm_cuda", "Matrix Multiply using CUDA");

  // Set parser value
  args::HelpFlag help(parser, "help", "Display this help menu", {'h', "help"});
  args::ValueFlag<int> widthA(parser, "widthA", "Width of matrix A", {"wA"}, 256);
  args::ValueFlag<int> widthB(parser, "widthB", "Width of matrix B", {"wB"}, 256);
  args::ValueFlag<int> heightA(parser, "heightA", "Height of matrix A", {"hA"},256);
  args::ValueFlag<int> heightB(parser, "heightB", "Height of matrix B", {"hB"}, 256);
  
  // Invoke parser
  try {
    parser.ParseCLI(argc, argv);
  } catch (args::Help) {
    std::cout << parser;
    return 0;
  } catch (args::ParseError e) {
    std::cerr << e.what() << std::endl;
    std::cerr << parser;
    return 1;
  } catch (args::ValidationError e) {
    std::cerr << e.what() << std::endl;
    std::cerr << parser;
    return 1;
  }

  // Initialize matrix dimensions
  int WA = args::get(widthA);
  int WB = args::get(widthB);
  int HA = args::get(heightA);
  int HB = args::get(heightB);
  int WC = WA;
  int HC = HB;

  // Setup CUDA environnement 
  hipError_t error;

  hipDeviceProp_t deviceProp;
  int devID = 0;
  error = hipGetDevice(&devID);

  if (error != hipSuccess) {
    printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
  }

  error = hipGetDeviceProperties(&deviceProp, devID);

  if (deviceProp.computeMode == hipComputeModeProhibited) {
    std::cerr << "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice() ." <<std::endl;
    exit(EXIT_SUCCESS);
  }

  if (error != hipSuccess) {
    printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
  } else {
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
  }

  // utilities
  hipEvent_t start;
  hipEvent_t stop;
  float msecTotal;

  // allocate host memory for matrices A and B
  unsigned int size_A = WA * HA;
  unsigned int mem_size_A = sizeof(float) * size_A;
  float *h_A = (float *)malloc(mem_size_A);
  unsigned int size_B = WB * HB;
  unsigned int mem_size_B = sizeof(float) * size_B;
  float *h_B = (float *)malloc(mem_size_B);
  
  // initialize host memory
  fill_random<REAL>(h_A, WA, HA);
  fill_random<REAL>(h_B, WB, HB);
 
  // allocate device memory
  float *d_A;
  hipMalloc((void **)&d_A, mem_size_A);
  float *d_B;
  hipMalloc((void **)&d_B, mem_size_B);

  // allocate device memory for result
  unsigned int size_C = WA * HB;
  unsigned int mem_size_C = sizeof(float) * size_C;
  float *d_C;
  hipMalloc((void **)&d_C, mem_size_C);

  // allocate host memory for the result
  float *h_C = (float *)malloc(mem_size_C);

  dim3 threads, grid;

  // create and start timer
  hipEventCreate(&start);
  hipEventRecord(start, NULL);
 
  // copy host memory to device
  hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

  // setup execution parameters
  threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
  grid = dim3(WC / threads.x, HC / threads.y);
  
  // execute the kernel
  gemm_naive<<<grid, threads>>>(d_C, d_A, d_B, WA, WB);

  // copy result from device to host
  hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

  // stop and destroy timer
  hipEventCreate(&stop);
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&msecTotal, start, stop);

  /* Performance computation, results and performance printing ------------ */
  auto flop = 2 * (float)WC * (float)HC * (float)WA;

  std::cout << " == Performances " << std::endl;
  std::cout << "\t Processing time: " << msecTotal << " (ms)"
            << std::endl;
  std::cout << "\t GFLOPS: " << flop / msecTotal / 1e+6 << std::endl;

  return (EXIT_SUCCESS);
}
