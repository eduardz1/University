// Assigns every element in an array with its index.
// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void simple(float *c) {
  c[threadIdx.x] = threadIdx.x;
  c[threadIdx.x] *= c[threadIdx.x];
}

int main() {
  // Define problem size
  const int N = 16;

  // Define number of blocks
  const int blocksize = 16;

  // Create host and device data strutures
  float *c_h = new float[N];
  float *c_d;

  // Give size of array to allocate on GPU
  const int size = N * sizeof(float);

  //	Allocate array on GP GPU
  hipMalloc((void **)&c_d, size);

  // Define workspace topology
  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);

  // Execute kernel
  simple<<<dimGrid, dimBlock>>>(c_d);

  // Wait for kernel completion
  hipDeviceSynchronize();

  // Copy result of computation back on host
  hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++)
    std::cout << c_h[i] << " ";

  std::cout << std::endl;

  // Free memory
  hipFree(c_d);
  delete[] c_h;

  std::cout << "done" << std::endl;

  return EXIT_SUCCESS;
}
