
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add_matrix(float *a, float *b, float *c, const int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < n && j < n) {
        c[i * n + j] = a[i * n + j] + b[i * n + j];
    }
}

int main() {
    const int N = 16;

    float *a, *b, *c;

    hipMallocManaged(&a, N * N * sizeof(float));
    hipMallocManaged(&b, N * N * sizeof(float));
    hipMallocManaged(&c, N * N * sizeof(float));

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            a[i + j * N] = 10 + i;
            b[i + j * N] = float(j) / N;
        }
    }

    dim3 dimBlock(16, 16);
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x, (N + dimBlock.y - 1) / dimBlock.y);

    add_matrix<<<dimGrid, dimBlock>>>(a, b, c, N);

    hipDeviceSynchronize();

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout<<c[i + j * N]<<" ";
        }
        std::cout<<std::endl;
    }

    hipFree(a);
    hipFree(b);
    hipFree(c);

    return EXIT_SUCCESS;
}