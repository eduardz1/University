
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add_matrix(float *a, float *b, float *c, const int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if (i < n && j < n) {
    c[i * n + j] = a[i * n + j] + b[i * n + j];
  }
}

int main() {
  const int N = 16;

  float *a, *b, *c;

  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMallocManaged(&a, N * N * sizeof(float));
  hipMallocManaged(&b, N * N * sizeof(float));
  hipMallocManaged(&c, N * N * sizeof(float));

  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      a[i + j * N] = 10 + i;
      b[i + j * N] = float(j) / N;
    }
  }

  dim3 dimBlock(16, 16);
  dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x,
               (N + dimBlock.y - 1) / dimBlock.y);

  hipEventRecord(start);
  add_matrix<<<dimGrid, dimBlock>>>(a, b, c, N);
  hipEventRecord(stop);

  hipDeviceSynchronize();

  hipEventSynchronize(start);
  hipEventSynchronize(stop);

  float time;
  hipEventElapsedTime(&time, start, stop);

  std::cout << "Time: " << time << " ms" << std::endl;

  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      std::cout << c[i + j * N] << " ";
    }
    std::cout << std::endl;
  }

  hipFree(a);
  hipFree(b);
  hipFree(c);

  return EXIT_SUCCESS;
}