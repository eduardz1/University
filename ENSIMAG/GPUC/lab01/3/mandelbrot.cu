#include "hip/hip_runtime.h"
#include "readppm.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_complex.h>
#include <iostream>
#include <string>

#define DIM 512
#define MAXITER 20

#define offsetx -200
#define offsety 0
#define zoom 0
#define scale 1.5

__global__ void mandelbrot(int *fractalValue, int x, int y) {
  float jx = scale * (float)(DIM / 2 - x + offsetx / scale) / (DIM / 2);
  float jy = scale * (float)(DIM / 2 - y + offsety / scale) / (DIM / 2);

  hipFloatComplex c = make_hipFloatComplex(jx, jy);
  hipFloatComplex a = make_hipFloatComplex(jx, jy);

  int i = 0;
  for (i = 0; i < MAXITER; i++) {
    a = hipCmulf(a, a);        // Multiply a by itself
    a = hipCaddf(a, c);        // Add c to the result
    if (hipCabsf(a) > 1000.0f) // Use hipCabsf to get the magnitude
      break;
  }

  *fractalValue = i;
}

void computeFractal(unsigned char *ptr) {
  // map from x, y to pixel position
  for (int x = 0; x < DIM; x++)
    for (int y = 0; y < DIM; y++) {
      int offset = x + y * DIM;

      // now calculate the value at that position
      int fractalValue;
      mandelbrot<<<1, 1>>>(&fractalValue, x, y);

      std::clog << "Fractal value: " << fractalValue << std::endl;

      hipDeviceSynchronize();

      // Colorize it
      int red = 255 * fractalValue / MAXITER;
      if (red > 255)
        red = 255 - red;
      int green = 255 * fractalValue * 4 / MAXITER;
      if (green > 255)
        green = 255 - green;
      int blue = 255 * fractalValue * 20 / MAXITER;
      if (blue > 255)
        blue = 255 - blue;

      ptr[offset * 4 + 0] = red;
      ptr[offset * 4 + 1] = green;
      ptr[offset * 4 + 2] = blue;

      ptr[offset * 4 + 3] = 255;

      hipFree(&fractalValue);
    }
}

// Main program, inits
int main(int argc, char **argv) {
  unsigned char *pixels;

  pixels = (unsigned char *)malloc(DIM * DIM * 4);

  computeFractal(pixels);

  // Dump to PPM
  writeppm("fractalout_gpu.ppm", DIM, DIM, pixels);
}